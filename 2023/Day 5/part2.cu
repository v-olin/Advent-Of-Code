
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <stdint.h>
 
#define IS_DIGIT(c) (c >= '0' && c <= '9')
#define BLOCKS 512
#define THREADS 512
 
bool InitCUDA()
{
    int count;
    hipGetDeviceCount(&count);
    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }
 
    int i;
    for (i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1)  {
                break;
            }
        }
    }
 
    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }
    hipSetDevice(i);
    return true;
}
 
size_t parseNum(std::string s, int64_t* num) {
    int64_t n = 0;
    size_t i = 0;
    while (IS_DIGIT(s[i])) {
        n *= 10;
        n += s[i] - '0';
        i++;
    }
    *num = n;
    return i;
}
 
struct range {
    int64_t dst = 0;
    int64_t src = 0;
    int64_t len = 0;
 
    range() {}
 
    range(int64_t dst, int64_t src, int64_t len) : dst(dst), src(src), len(len) {}
 
    range(std::string s) {
        size_t pos = 0;
        pos += parseNum(s.substr(pos), &dst) + 1;
        pos += parseNum(s.substr(pos), &src) + 1;
        pos += parseNum(s.substr(pos), &len);
    }
};
 
struct map {
    range *ranges;
    unsigned int len;
};
 
__global__ static void kernel(int64_t start, int64_t range, int64_t** maps, unsigned int* mapLens, int64_t* locs) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
 
    int64_t min = INT64_MAX;
    for (int64_t i = start + idx; i < start + range; i += stride) {
        int64_t loc = i;
        for (int m = 0; m < 7; m++) {
            for (int k = 0; k < mapLens[m]; k++) {
                int64_t dst = maps[m][k * 3];
                int64_t src = maps[m][k * 3 + 1];
                int64_t len = maps[m][k * 3 + 2];
 
                if (loc >= src && loc <= src + len) {
                    loc = dst + (loc - src);
                    break;
                }
            }
        }
 
        if (loc < min) {
            min = loc;
        }
    }
 
    if (min < INT64_MAX) {
        locs[idx] = min;
    }
}
 
void parseSeeds(std::vector<int64_t>& seeds, std::string s) {
    size_t pos = 7;
    while (pos < s.size()) {
        int64_t n;
        pos += parseNum(s.substr(pos), &n);
        seeds.push_back(n);
        pos++;
    }
}
 
void parseRanges(std::vector<std::vector<range>>& maps, std::vector<std::string>& lines) {
    size_t mapNum = 0;
    std::vector<range> m;
    std::string s;
 
    for (size_t i = 1; i < lines.size(); i++) {
        s = lines[i];
        if (IS_DIGIT(s[0])) {
            m.push_back(range(s));
        }
        else {
            maps.push_back(m);
            m = std::vector<range>();
        }
    }
 
    maps.push_back(m);
}
 
int64_t runKernel(std::vector<int64_t>& seeds, std::vector<std::vector<range>>& maps) {
    size_t numMaps = maps.size();
 
    unsigned int* mapLens;
    hipMallocManaged(&mapLens, sizeof(unsigned int) * numMaps);
    for (size_t i = 0; i < numMaps; i++) {
        mapLens[i] = maps[i].size();
    }
 
    int64_t** maps_d;
    hipMallocManaged(&maps_d, sizeof(int64_t*) * numMaps);
    for (size_t i = 0; i < numMaps; i++) {
        hipMallocManaged(&maps_d[i], sizeof(int64_t) * mapLens[i] * 3);
        for (size_t j = 0; j < mapLens[i]; j++) {
            maps_d[i][j * 3] = maps[i][j].dst;
            maps_d[i][j * 3 + 1] = maps[i][j].src;
            maps_d[i][j * 3 + 2] = maps[i][j].len;
        }
    }
 
    int64_t totMin = INT64_MAX;
    for (size_t i = 0; i < seeds.size(); i += 2) {
        int64_t start = seeds.at(i);
        int64_t end = seeds.at(i + 1);
 
        int64_t* locs;
        hipMallocManaged(&locs, sizeof(int64_t) * BLOCKS * THREADS);
        for (size_t i = 0; i <  BLOCKS * THREADS; i++) {
            locs[i] = INT64_MAX;
        }
 
        kernel<<<BLOCKS, THREADS>>>(start, end, maps_d, mapLens, locs);
        hipDeviceSynchronize();
 
        int64_t min = INT64_MAX;
        for (size_t j = 0; j < BLOCKS * THREADS; j++) {
            if (locs[j] < min) {
                min = locs[j];
            }
        }
 
        if (min < totMin) {
            totMin = min;
        }
 
        hipFree(locs);
    }
 
    for (size_t i = 0; i < numMaps; i++) {
        hipFree(maps_d[i]);
    }
 
    hipFree(maps_d);
    hipFree(mapLens);
 
    return totMin;
}
 
int main(int argc, char** argv) {
    if (argc != 2) {
        std::cout << "No input file" << std::endl;
        return -1;
    }
 
    if (!InitCUDA()) {
        return -1;
    }
 
    std::ifstream input(argv[1]);
    std::string s;
    std::vector<std::string> lines;
    std::vector<int64_t> seeds;
    std::vector<std::vector<range>> maps;
 
    getline(input, s);
    parseSeeds(seeds, s);
 
    while (getline(input, s)) {
        if (!s.empty())
            lines.push_back(s);
    }
 
    input.close();
 
    parseRanges(maps, lines);
 
    int64_t part2 = runKernel(seeds, maps);
 
    std::cout << "Part 2: " << part2 << std::endl;
    return 0;
}
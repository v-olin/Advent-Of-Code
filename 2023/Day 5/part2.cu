
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <stdint.h>

#define IS_DIGIT(c) (c >= '0' && c <= '9')
#define ULL unsigned long long
#define BLOCKS 512
#define THREADS 512

bool InitCUDA()
{
    int count;
    hipGetDeviceCount(&count);
    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;
    for (i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1)  {
                break;
            }
        }
    }
    
    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }
    hipSetDevice(i);
    return true;
}

size_t parseNum(std::string s, uint64_t* num) {
    uint64_t n = 0;
    size_t i = 0;
    while (IS_DIGIT(s[i])) {
        n *= 10;
        n += s[i] - '0';
        i++;
    }
    *num = n;
    return i;
}

struct range {
    uint64_t dst = 0;
    uint64_t src = 0;
    uint64_t len = 0;

    range() {}

    range(uint64_t dst, uint64_t src, uint64_t len) : dst(dst), src(src), len(len) {}

    range(std::string s) {
        size_t pos = 0;
        pos += parseNum(s.substr(pos), &dst) + 1;
        pos += parseNum(s.substr(pos), &src) + 1;
        pos += parseNum(s.substr(pos), &len);
    }
};

struct map {
    range *ranges;
    unsigned int len;
};

__global__ static void kernel(ULL start, ULL end, ULL** maps, unsigned int* mapLens, ULL* locs) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    ULL min = UINT64_MAX;
    for (ULL i = start + idx; i <= end; i += stride) {
        ULL loc = i;
        for (int m = 0; m < 7; m++) {
            for (int k = 0; k < mapLens[m]; k++) {
                ULL dst = maps[m][k * 3];
                ULL src = maps[m][k * 3 + 1];
                ULL len = maps[m][k * 3 + 2];

                if (loc >= src && loc < src + len) {
                    loc = dst + loc - src;
                    break;
                }
            }
        }

        if (loc < min) {
            min = loc;
        }
    }

    if (min < UINT64_MAX) {
        locs[idx] = min;
    }
}

void parseSeeds(std::vector<uint64_t>& seeds, std::string s) {
    size_t pos = 7;
    while (pos < s.size()) {
        uint64_t n;
        pos += parseNum(s.substr(pos), &n);
        seeds.push_back(n);
        pos++;
    }
}

void parseRanges(std::vector<std::vector<range>>& maps, std::vector<std::string>& lines) {
    size_t mapNum = 0;
    std::vector<range> m;
    std::string s;

    for (size_t i = 1; i < lines.size(); i++) {
        s = lines[i];
        if (IS_DIGIT(s[0])) {
            m.push_back(range(s));
        }
        else {
            maps.push_back(m);
            m = std::vector<range>();
        }
    }

    maps.push_back(m);
}

ULL runKernel(std::vector<uint64_t>& seeds, std::vector<std::vector<range>>& maps) {
    size_t numMaps = maps.size();

    unsigned int* mapLens;
    hipMallocManaged(&mapLens, sizeof(unsigned int) * numMaps);
    for (size_t i = 0; i < numMaps; i++) {
        mapLens[i] = maps[i].size();
    }

    ULL** maps_d;
    hipMallocManaged(&maps_d, sizeof(ULL*) * numMaps);
    for (size_t i = 0; i < numMaps; i++) {
        hipMallocManaged(&maps_d[i], sizeof(ULL) * mapLens[i] * 3);
        for (size_t j = 0; j < mapLens[i]; j++) {
            maps_d[i][j * 3] = maps[i][j].dst;
            maps_d[i][j * 3 + 1] = maps[i][j].src;
            maps_d[i][j * 3 + 2] = maps[i][j].len;
        }
    }

    ULL totMin = UINT64_MAX;
    for (size_t i = 0; i < seeds.size(); i += 2) {
        ULL start = seeds.at(i);
        ULL end = seeds.at(i + 1);

        ULL* locs;
        hipMallocManaged(&locs, sizeof(ULL) * BLOCKS * THREADS);

        kernel<<<BLOCKS, THREADS>>>(start, end, maps_d, mapLens, locs);
        hipDeviceSynchronize();

        ULL min = UINT64_MAX;
        for (size_t j = 0; j < BLOCKS * THREADS; j++) {
            if (locs[j] < min) {
                min = locs[j];
            }
        }

        if (min < totMin) {
            totMin = min;
        }

        hipFree(locs);
    }

    for (size_t i = 0; i < numMaps; i++) {
        hipFree(maps_d[i]);
    }

    hipFree(maps_d);
    hipFree(mapLens);

    return totMin;
}

int main(int argc, char** argv) {
    if (argc != 2) {
        std::cout << "No input file" << std::endl;
        return -1;
    }

    if (!InitCUDA()) {
        return -1;
    }

    std::ifstream input(argv[1]);
    std::string s;
    std::vector<std::string> lines;
    std::vector<uint64_t> seeds;
    std::vector<std::vector<range>> maps;

    getline(input, s);
    parseSeeds(seeds, s);

    while (getline(input, s)) {
        if (!s.empty())
            lines.push_back(s);
    }
    
    input.close();

    parseRanges(maps, lines);

    std::cout << "Seeds: " << seeds.size() << std::endl;
    std::cout << "Maps: " << maps.size() << std::endl;

    for (size_t i = 0; i < maps.size(); i++) {
        std::cout << "Map " << i << ": " << maps[i].size() << std::endl;
    }

    ULL part2 = runKernel(seeds, maps);
    
    std::cout << "Part 2: " << part2 << std::endl;
    return 0;
}